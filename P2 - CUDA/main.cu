#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>


#define WIDTH 3833
#define HEIGHT 2160

bool checkResults(uchar4* rgba, uchar3* bgr, int size) {

    bool correct = true;

    for (int i=0; i < size; ++i) {
	// In case you want to see actual values
	if (i==3) {
		unsigned char x, y, z, w;
		x = rgba[i].x;
		y = rgba[i].y;
		z = rgba[i].z;
		w = rgba[i].w;
		std::cout << "First position x=" << (unsigned int)x << " y=" << (unsigned int)y << " z=" << (unsigned int)z << " w=" << (unsigned int)w << std::endl;
	}
        correct &= rgba[i].x == bgr[i].z;
        correct &= rgba[i].y == bgr[i].y;
        correct &= rgba[i].z == bgr[i].x;
        correct &= rgba[i].w == 255;
    }

    return correct;
}

__global__ void convertBGR2RGBA_1for(uchar3 *bgr, uchar4* rgba, int width, int height) {

	//int position = 0; // 0 is not correct. Compute each thread position;
	int position = threadIdx.x + blockIdx.x * blockDim.x;

	//printf("GPU - i = %d, j = %d\n", positionx, positiony);
	// Protection to avoid segmentation fault
	if (position < width * height) {	
		rgba[position].x = bgr[position].z;
		rgba[position].y = bgr[position].y;
		rgba[position].z = bgr[position].x;
		rgba[position].w = 255;
	}
}

__global__ void convertBGR2RGBA_2for(uchar3 *bgr, uchar4* rgba, int width, int height) {

	//int position = 0; // 0 is not correct. Compute each thread position;
	int positionx = threadIdx.x + blockIdx.x * blockDim.x;
	int positiony = threadIdx.y + blockIdx.y * blockDim.y;
	//int position = positiony * WIDTH + positionx;
	int position = positionx * HEIGHT + positiony;

	//printf("GPU - i = %d, j = %d\n", positionx, positiony);
	// Protection to avoid segmentation fault
	if (positionx < width ||  positiony < height) {
		rgba[position].x = bgr[position].z;
		rgba[position].y = bgr[position].y;
		rgba[position].z = bgr[position].x;
		rgba[position].w = 255;
	}
}

__global__ void convertBGR2RGBA_optBasic(uchar3 *bgr, uchar4* rgba, int width, int height) {

	int position = threadIdx.x + blockIdx.x * blockDim.x;
	uchar3 tempbgr = bgr[position];
	uchar4 temprgba;
	// Protection to avoid segmentation fault
	if (position < width * height) {	
		temprgba.x = tempbgr.z;
		temprgba.y = tempbgr.y;
		temprgba.z = tempbgr.x;
		temprgba.w = 255;
		rgba[position] = temprgba;
	}
}

__global__ void convertBGR2RGBA_optBasic2(uchar3 *bgr, uchar4* rgba, int width, int height) {

	int position = 2*(threadIdx.x + blockIdx.x * blockDim.x);

    if (position < width * height) {
        rgba[position+0].x = bgr[position+0].z;
        rgba[position+1].x = bgr[position+1].z;

        rgba[position+0].y = bgr[position+0].y;
        rgba[position+1].y = bgr[position+1].y;

        rgba[position+0].z = bgr[position+0].x;
        rgba[position+1].z = bgr[position+1].x;

        rgba[position+0].w = 255;
        rgba[position+1].w = 255;
    }
}


__global__ void convertBGR2RGBA_shared(uchar3 *bgr, uchar4* rgba, int width, int height) {

	extern __shared__ uchar3 shared_bgr[1024];
	extern __shared__ uchar4 shared_rgba[1024];

    int tid = threadIdx.x;

    int position = threadIdx.x + blockIdx.x * blockDim.x;

    if (position < width * height) {

        shared_bgr[tid] = bgr[position];

    	__syncthreads();

        shared_rgba[tid].x = shared_bgr[tid].z;
	    shared_rgba[tid].y = shared_bgr[tid].y;
	    shared_rgba[tid].z = shared_bgr[tid].x;
	    shared_rgba[tid].w = 255;

    	__syncthreads();
    	rgba[position] = shared_rgba[tid];
	}
}

int main() {

    uchar3 *h_bgr, *d_bgr;
    uchar4 *h_rgba, *d_rgba;

    int bar_widht = HEIGHT/3;

    // Alloc and generate BGR bars.
    h_bgr = (uchar3*)malloc(sizeof(uchar3)*WIDTH*HEIGHT);
    for (int i=0; i < WIDTH * HEIGHT; ++i) {
        if (i < bar_widht) {
		uchar3 temp = {255, 0, 0};
		h_bgr[i] = temp; 
	} else if (i < bar_widht*2) {
		uchar3 temp = {0, 255, 0};
		h_bgr[i] = temp;
	} else { 
		uchar3 temp = {0, 0, 255};
		h_bgr[i] = temp;
	}
    }

    // Alloc RGBA pointers
    h_rgba = (uchar4*)malloc(sizeof(uchar4)*WIDTH*HEIGHT);

    // Alloc gpu pointers
    hipError_t error = hipMalloc(&d_bgr, sizeof(uchar3) * WIDTH * HEIGHT);
    if (error != hipSuccess) {
	std::cout << "Error in hipMalloc" << std::endl;
    }

    error = hipMalloc(&d_rgba, sizeof(uchar4) * WIDTH * HEIGHT);
    if (error != hipSuccess) {
	std::cout << "Error in hipMalloc" << std::endl;
    }
    
    // Copy data to GPU
    error = hipMemcpy(d_bgr, h_bgr, sizeof(uchar3) * WIDTH * HEIGHT, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
	std::cout << "Error in hipMemcpy" << std::endl;
    }

    // Init output buffer to 0
    error = hipMemset(d_rgba, 0, WIDTH*HEIGHT*sizeof(uchar4));
    if (error != hipSuccess) {
        std::cout << "Error in hipMemset" << std::endl;
    }

    //FUNCIO AMB UN SOL FOR
    /*dim3 block(64, 1, 1);
    dim3 grid(ceil(WIDTH*HEIGHT/(float)block.x), 1, 1);
    convertBGR2RGBA_1for<<<grid, block, 0, 0>>>(d_bgr, d_rgba, WIDTH, HEIGHT);*/

    //FUNCIO AMB DOS FORS
    /*dim3 block(8, 8, 1);
    dim3 grid(ceil(WIDTH/(float)block.x),ceil(HEIGHT/(float)block.y), 1);
    convertBGR2RGBA_2for<<<grid, block, 0, 0>>>(d_bgr, d_rgba, WIDTH, HEIGHT);*/


    //OPTIMITZACIONS DE MEMORIA
    dim3 block(128, 1, 1);
    dim3 grid(ceil(WIDTH*HEIGHT/(float)block.x),1, 1);
    convertBGR2RGBA_optBasic<<<grid, block, 0, 0>>>(d_bgr, d_rgba, WIDTH, HEIGHT);

    //OPTIMITZACIO DE MEMORIA 2
    /*dim3 block(512, 1, 1);
    dim3 grid(ceil(WIDTH*HEIGHT/(float)block.x),1, 1);
    convertBGR2RGBA_optBasic2<<<grid, block, 0, 0>>>(d_bgr, d_rgba, WIDTH, HEIGHT);*/

    //SHARED MEMORY
    /*dim3 block(1024, 1, 1);
    dim3 grid(ceil(WIDTH*HEIGHT/(float)block.x), 1, 1);
    convertBGR2RGBA_shared<<<grid, block>>>(d_bgr, d_rgba, WIDTH, HEIGHT);*/

    hipDeviceSynchronize();

    // Copy data back from GPU to CPU
    error = hipMemcpy(h_rgba, d_rgba, sizeof(uchar4) * WIDTH * HEIGHT, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
	std::cout << "Error in hipMemcpy." << std::endl;
	std::cout << hipGetErrorString(error) << std::endl;
    }

    // Check results
    bool ok = checkResults(h_rgba, h_bgr, WIDTH*HEIGHT);
    if (ok) {
        std::cout << "Executed!! Results OK." << std::endl;
    } else {
        std::cout << "Executed!! Results NOT OK." << std::endl;
    }

    // Free CPU pointers
    free(h_rgba);
    free(h_bgr);

    // Free cuda pointers
    error = hipFree(d_bgr);
    if (error != hipSuccess) {
	std::cout << "Error in hipFree" << std::endl;
	std::cout << hipGetErrorString(error) << std::endl;
    }
    error = hipFree(d_rgba);
    if (error != hipSuccess) {
	std::cout << "Error in hipFree" << std::endl;
	std::cout << hipGetErrorString(error) << std::endl;
    }

    // Clean GPU device
    error = hipDeviceReset();
    if (error != hipSuccess) {
	std::cout << "Error in hipDeviceReset" << std::endl;
    }

    return 0;

}
